
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
#include <time.h> 

using namespace std;

// ������� ��������� ������������
void simpleMatMul(int* c, int* a, int* b, int rows1, int cols1, int cols2) {
    for (unsigned int i = 0; i < rows1; i++)
    {
        for (unsigned int j = 0; j < cols2; j++)
        {
            c[i * cols2 + j] = 0;
            for (unsigned int k = 0; k < cols1; k++)
            {
                c[i * cols2 + j] += a[i * cols1 + k] * b[k * cols2 + j];
            }
        }
    }
}

__global__ void matMulKernel(int* c, int* a, int* b, int rows1, int cols1, int cols2)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= rows1 || j >= cols2)
    {
        return;
    }
    c[i * cols2 + j] = 0;
    for (int k = 0; k < cols1; k++)
    {
        c[i * cols2 + j] += a[i * cols1 + k] * b[k * cols2 + j];
    }
}
// Helper function for using CUDA to add vectors in parallel.
hipError_t matMulWithCuda(int* c, int* a, int* b, int rows1, int cols1, int cols2)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, rows1 * cols2 * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

    cudaStatus = hipMalloc((void**)&dev_a, rows1 * cols1 * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

    cudaStatus = hipMalloc((void**)&dev_b, cols1 * cols2 * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, rows1 * cols1 * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

    cudaStatus = hipMemcpy(dev_b, b, cols1 * cols2 * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

    // ������ ����� �������� �������, �� � �������� ����� ��������� Cuda, � ���, ��� 1-2 ������ ����� �� ����� ��������(�� ���� ��� ���� ������ ������� if)
    dim3 blockSize = dim3(32, 32, 1);
    dim3 gridSize = dim3(rows1 / 32 + 1, cols1 / 32 + 1, 1);
    // Launch a kernel on the GPU with one thread for each element.
    matMulKernel <<< gridSize, blockSize >>> (dev_c, dev_a, dev_b, rows1, cols1, cols2);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "matMulKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching matMulKernel!\n", cudaStatus);
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, rows1 * cols2 * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}


int main()
{
    int rows1;
    int cols1;
    int cols2;

    cout << "Enter the number of rows and columns" << endl;
    cout << "Number of rows for 1 matrix:" << endl;
    cin >> rows1; 
    cout << "Number of columns for 1 matrix:" << endl;
    cin >> cols1;
    cout << "Number of columns for 2 matrix:" << endl;
    cin >> cols2;

    // �������� ������ � ��������� �������
    int* a = new int[rows1 * cols1];
    int* b = new int[cols1 * cols2];
    int* c = new int[rows1 * cols2];

    for (int i = 0; i < rows1; i++)
    {
        for (int j = 0; j < cols1; j++)
        {
            a[i * cols1 + j] = i * cols1 + j;
        }
    }
    for (int i = 0; i < cols1; i++)
    {
        for (int j = 0; j < cols2; j++)
        {
            b[i * cols2 + j] = i * cols2 + j;
        }
    }

    clock_t start = clock();
    // Add vectors in parallel.
    hipError_t cudaStatus = matMulWithCuda(c, a, b, rows1, cols1, cols2);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "matMulWithCuda failed!");
    }
    clock_t end = clock();
    cout << "Cuda time: " << (double)(end - start) / CLOCKS_PER_SEC << endl;

    c = new int[rows1 * cols2];

    start = clock();
    simpleMatMul(c, a, b, rows1, cols1, cols2);
    end = clock();
    cout << "CPU time: " << (double)(end - start) / CLOCKS_PER_SEC << endl;

    /*for (int i = 0; i < rows1; i++)
    {
        for (int j = 0; j < cols1; j++)
        {
            cout << a[i * cols1 + j] << ' ';
        }
        cout << endl;
    }
    for (int i = 0; i < cols1; i++)
    {
        for (int j = 0; j < cols2; j++)
        {
            cout << b[i * cols2 + j] << ' ';
        }
        cout << endl;
    }
    for (int i = 0; i < rows1; i++)
    {
        for (int j = 0; j < cols2; j++)
        {
            cout << c[i * cols2 + j] << ' ';
        }
        cout << endl;
    }*/
    delete[] a;
    delete[] b;
    delete[] c;

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

