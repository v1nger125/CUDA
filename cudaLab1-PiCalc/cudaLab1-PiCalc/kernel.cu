
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
#include <time.h> 
#include <random>

using namespace std;

// �������� ������ �������� ������ �� ������ nvidia http://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
// �� ����� ����������� ����������� � ������ ��������, �� � �� ����� ������
__global__ void piCalcKernel(int *d_odata, double* a, double* b, int size)
{
    __shared__ int sdata[1024];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid] = ((a[i] * a[i] + b[i] * b[i]) < 1 ? 1 : 0);

    __syncthreads();
    for (int j = blockDim.x/2; j > 0; j>>=1)
    {
        if (tid < j)
        {
            sdata[tid] += sdata[tid + j];
        }
        __syncthreads();
    }
    if (tid == 0) d_odata[blockIdx.x] = sdata[0];
}
// Helper function for using CUDA to add vectors in parallel.
hipError_t piCalcWithCuda(int* odata, double* a, double* b, int size)
{
    double* dev_a = 0;
    double* dev_b = 0;
    hipError_t cudaStatus;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

    int* dev_odata = 0;
    cudaStatus = hipMalloc((void**)&dev_odata, (size / 1024) * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }


    dim3 blockSize = dim3(1024, 1, 1);
    dim3 gridSize = dim3(size/1024, 1, 1);

    // Launch a kernel on the GPU with one thread for each element.
    piCalcKernel <<<gridSize, blockSize >>> (dev_odata, dev_a, dev_b, size);
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "piCalcKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching piCalcKernel!\n", cudaStatus);
    }
    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(odata, dev_odata, (size / 1024) * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_odata);

    return cudaStatus;
}

double simplePiCalc(double* a, double* b, int size) {
    int sum = 0;
    for (int i = 0; i < size; i++)
    {
        if (a[i] * a[i] + b[i] * b[i] < 1) 
        {
            sum++;
        }
    }
    return (double) 4*sum / size;
}
int main()
{
    int N;

    cout << "Enter the number of numbers: " << endl;
    cin >> N;

    // �������� ������ � ���������� �����
    double* a = new double[N];
    double* b = new double[N];
    int* cudaOut = new int[(N / 1024)];

    std::default_random_engine generator;
    std::uniform_real_distribution<double> distribution(-1.0, 1.0);
    for (int i = 0; i < N; i++)
    {
        a[i] = distribution(generator);
        b[i] = distribution(generator);
    }

    
    // cuda ����������
    double result = 0;
    clock_t start = clock();
    hipError_t cudaStatus = piCalcWithCuda(cudaOut, a, b, N);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "piCalcWithCuda failed!");
    }
    for (int i = 0; i < (N / 1024); i++)
    {
       result += cudaOut[i];
    }
    result = 4 * (double) result / (N - N%1024);
    clock_t end = clock();
    cout << "Cuda time: " << (double)(end - start) / CLOCKS_PER_SEC << endl;
    cout << "Cuda result: " << result << endl;
    // cpu ����������
    result = 0;
    start = clock();
    result = simplePiCalc(a, b, N);
    end = clock();
    cout << "CPU time: " << (double)(end - start) / CLOCKS_PER_SEC << endl;
    cout << "CPU result: " << result << endl;
    delete[] cudaOut;
    delete[] a;
    delete[] b;

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

